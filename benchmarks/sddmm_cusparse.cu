
#include <hip/hip_runtime.h>
#include <cstdint>
#include <hipsparse.h>

hipsparseHandle_t handle;
const float alpha = 1.0;
const float beta = 0.0;
hipsparseSpMatDescr_t A;
hipsparseConstDnMatDescr_t C;
hipsparseConstDnMatDescr_t D;
size_t *ext_buffer;

extern "C"
void cusparse_init_handle() {
  hipsparseCreate(&handle);
}

extern "C"
int sddmm_init(
  int64_t *A_row_indices, int64_t *A_col_indices, float *A_values,
  float *C_data, float *D_data, int64_t N, int64_t M, int64_t K, int64_t nnz
) {
  hipsparseCreateCsr(
    &A, N, M, nnz, A_row_indices, A_col_indices, A_values, HIPSPARSE_INDEX_64I,
    HIPSPARSE_INDEX_64I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F
  );
  hipsparseCreateConstDnMat(&C, N, K, K, C_data, HIP_R_32F, HIPSPARSE_ORDER_ROW);
  hipsparseCreateConstDnMat(&D, K, M, M, D_data, HIP_R_32F, HIPSPARSE_ORDER_ROW);
  size_t buffer_size;
  hipsparseSDDMM_bufferSize(
    handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE,
    &alpha, C, D, &beta, A, HIP_R_32F, HIPSPARSE_SDDMM_ALG_DEFAULT, &buffer_size
  );
  hipError_t err = hipMalloc(&ext_buffer, buffer_size);
  if (err == hipErrorOutOfMemory) {
    hipsparseDestroySpMat(A);
    hipsparseDestroyDnMat(C);
    hipsparseDestroyDnMat(D);
    return 1;
  }
  hipsparseSDDMM_preprocess(
      handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE,
      &alpha, C, D, &beta, A, HIP_R_32F, HIPSPARSE_SDDMM_ALG_DEFAULT, ext_buffer
  );
  return 0;
}

extern "C"
void sddmm_deinit() {
  hipsparseDestroySpMat(A);
  hipsparseDestroyDnMat(C);
  hipsparseDestroyDnMat(D);
  hipFree(ext_buffer);
}

__global__
void custom_inplace_elemwise_mul(float *A, float *B, int64_t nnz) {
  int64_t i = blockIdx.x * 1024 + threadIdx.x;
  if (i < nnz) {
    A[i] *= B[i];
  }
}

extern "C"
int sddmm(float *A_values, float *B_values, int64_t nnz) {
  hipsparseSDDMM(
      handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE,
      &alpha, C, D, &beta, A, HIP_R_32F, HIPSPARSE_SDDMM_ALG_DEFAULT, ext_buffer
  );
  const int tpb = 1024;
  const int blocks = (nnz + tpb - 1) / tpb;
  custom_inplace_elemwise_mul<<<blocks, tpb>>>(A_values, B_values, nnz);
  return 0;
}
